
#include <hip/hip_runtime.h>
#include <cstdio>
#include <stdint.h>
#include <stdio.h>

#define MAX_THREADS (128)
#define WARP_SIZE (32)

struct CudaFormatMat {
  int32_t rows;
  int32_t cols;
  float *bytes;
};
struct YoloDetectionCuda {
  double confidence;
  double x;
  double y;
  double width;
  double height;
  int32_t class_id;
};

__forceinline__ __device__ float adjust_base(uintptr_t idx, float const factor,
                                             float const *row_bytes) {
  return row_bytes[idx] * factor;
}

__forceinline__ __device__ float x_adjust(uintptr_t idx, float const factor,
                                          float const *row_bytes) {
  return (adjust_base(idx, factor, row_bytes) / 640.0) * 800.0;
}

__forceinline__ __device__ float y_adjust(uintptr_t idx, float const factor,
                                          float const *row_bytes) {
  return (adjust_base(idx, factor, row_bytes) / 640.0) * 600.0;
}

__global__ void process_net(const uintptr_t num_rows, const uintptr_t num_cols,
                            const float threshold, const float factor,
                            const float *__restrict__ mat_bytes,
                            YoloDetectionCuda *__restrict__ processed_detects,
                            bool *__restrict__ processed_valid) {
  const auto id = blockIdx.x * blockDim.x + threadIdx.x;

  // Get rid of leftover threads
  if (id >= num_rows)
    return;

  float const *__restrict__ row = mat_bytes + (id * num_cols);

  const float confidence = row[4];
  const bool valid = confidence > threshold;
  processed_valid[id] = valid;

  // Skip remaining processing for invalid
  if (!valid)
    return;

  // Start at offset in data, then shift to starting at 0.
  uintptr_t class_id = 5;
  float class_value = row[class_id];
  for (uintptr_t i = 6; i < num_cols; ++i) {
    if (class_value < row[i]) {
      class_id = i;
      class_value = row[i];
    }
  }
  class_id -= 5;

  const float center_x = x_adjust(0, factor, row);
  const float center_y = y_adjust(1, factor, row);
  const float width = x_adjust(2, factor, row);
  const float height = y_adjust(3, factor, row);

  const float left = center_x - (width / 2.0);
  const float top = center_y - (height / 2.0);

  processed_detects[id] = YoloDetectionCuda{
      confidence, left, top, width, height, static_cast<int32_t>(class_id)};
}

extern "C" {
int process_net_kernel(CudaFormatMat *const result, uintptr_t const num_levels,
                       float const threshold, float const factor,
                       uintptr_t const total_rows,
                       YoloDetectionCuda *processed_detects,
                       bool *processed_valid) {

  /*
cudaStream_t kernel_stream;
cudaStreamCreate(&kernel_stream);
*/

  YoloDetectionCuda *processed_detects_cuda;
  bool *processed_valid_cuda;
  hipMalloc(&processed_detects_cuda, sizeof(YoloDetectionCuda) * total_rows);
  hipMalloc(&processed_valid_cuda, sizeof(bool) * total_rows);
  /*

  uintptr_t row_offset = 0;
  for (uintptr_t i = 0; i < num_levels; ++i) {
  CudaFormatMat *mat = result + i;
  auto num_rows = mat->rows;
  uintptr_t num_cols = static_cast<uintptr_t>(mat->cols);
  auto mat_size = num_rows * num_cols * sizeof(float);
  float *mat_bytes;

  cudaMalloc(&mat_bytes, mat_size);
  cudaMemcpy(mat_bytes, mat->bytes, mat_size, cudaMemcpyHostToDevice);

  int32_t blocksize = MAX_THREADS;
  int32_t block_count;
  if (num_rows < blocksize) {
  blocksize = num_rows;
  block_count = 1;
  } else {
  // Ceiling divide, from https://stackoverflow.com/a/14878734
  block_count = num_rows / MAX_THREADS + (num_rows % MAX_THREADS != 0);
  }

  process_net<<<block_count, blocksize, 0, kernel_stream>>>(
    num_rows, num_cols, threshold, factor, mat_bytes,
    processed_detects_cuda + row_offset, processed_valid_cuda + row_offset);

  cudaStreamSynchronize(kernel_stream);
  cudaFree(mat_bytes);

  row_offset += num_rows;
  }

  cudaMemcpy(processed_detects, processed_detects_cuda,
         sizeof(YoloDetectionCuda) * total_rows, cudaMemcpyDeviceToHost);
  cudaMemcpy(processed_valid, processed_valid_cuda, sizeof(bool) * total_rows,
         cudaMemcpyDeviceToHost);
             */
  hipFree(processed_detects_cuda);
  hipFree(processed_valid_cuda);
  /*
  cudaStreamDestroy(kernel_stream);
  */

  return 0;
}
}
